﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>


__device__ __managed__ u16 modify_time = 0;
__device__ __managed__ u16 create_time = 0;

__device__ __managed__ u16 most_adv_0_block = 0; //indicate the first available block
__device__ __managed__ u16 most_adv_0_fcb = 0; //indicate the first fcb which is not used





__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)
{
  // init variables
  fs->volume = volume;
  

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;

}





__device__ u32 fs_open(FileSystem *fs, char *s, int op)
{
	/* Implement open operation here */
	u32 location = 0;
	
	
	
	//printf("most_adv_0_block: %d; most_adv_0_fcb: %d\n", most_adv_0_block, most_adv_0_fcb);
	bool find = false; //indicate whether file name is found or not
	int i;
	for (i = 0; i < 1024; i++) {
		for (int j = 0; j < 20; j++) {
			if (fs->volume[4096 + i * 32 + j] == s[j]) {
				if (s[j] == '\0') {
					find = true;
					break;
				}
			}
			else break;
		}
		if (find) {
			break;
		}
		
	}
	///printf("find: %d\n",find);
	///printf("i: %d\n", i);
	if (find) {
		if (op == G_READ) {
			
			fs->volume[4096 + i * 32 + 20] = G_READ; //update mode
			location = fs->volume[4096 + i * 32 + 21] * 256 + fs->volume[4096 + i * 32 + 22]; //return location
		}
		else if (op == G_WRITE) {
			u16 ctime = fs->volume[4096 + i * 32 + 27] * 256 + fs->volume[4096 + i * 32 + 28];
			u16 size = fs->volume[4096 + i * 32 + 23] * 256 + fs->volume[4096 + i * 32 + 24];
			///printf("ttqqsize/256: %d; size!256: %d\n", fs->volume[4096 + i * 32 + 23], fs->volume[4096 + i * 32 + 24]);
			u16 block_offset = 0;
			if (size % 32 > 0) {
				block_offset = size / 32 + 1;
			}
			else if (size % 32 == 0) {
				block_offset = size / 32;
			}
			//u16 location_local = fs->volume[4096 + i * 32 + 21] * 256 + fs->volume[4096 + i * 32 + 22];
			// compact FCB
			/*for (int j = 4096 + (i + 1) * 32; j <= 4096 + ( most_adv_0_fcb - 2 )* 32; j += 32) {
				for (int k = 0; k < 32; k++) {
					fs->volume[j - 32 + k] = fs->volume[j + k];
				}

			}*/
			for (int j = 4096 + (i + 1) * 32; j < 4096 + most_adv_0_fcb * 32; j ++) {
				
					fs->volume[j - 32] = fs->volume[j];
				

			}
			most_adv_0_fcb--;
			//compact the volume
			u16 original_location = fs->volume[4096 + i * 32 + 21] * 256 + fs->volume[4096 + i * 32 + 22];
			//printf("original_location: %d\n", original_location);
			//u16 new_location = original_location - block_offset;
			/*for (int m = 36864 + (original_location - 1) * 32; m <= 36864 + (most_adv_0_block - 2) * 32; m += 32) {
				for (int n = 0; n < block_offset * 32; n++) {
					fs->volume[36864 + new_location + n] = fs->volume[36864 + original_location + n];
				}
			}*/
			for (int m = 36864 + original_location * 32; m < 36864 + most_adv_0_block * 32; m ++) {
				
					fs->volume[m-block_offset*32] = fs->volume[m];
				
			}
			most_adv_0_block -= block_offset;
			int l = i;
			// update location in FCB 
			for (l = 4096+i*32; l <= 4096 + (most_adv_0_fcb - 1) * 32; l += 32) {
				u16 new_location = fs->volume[l+ 21] * 256 + fs->volume[l + 22]-block_offset;
				fs->volume[l+ 21] = new_location / 256;
				fs->volume[l+ 22] = new_location % 256;
				//printf("location/256: %d; location!256: %d\n", fs->volume[4096 + l * 32 + 21], fs->volume[4096 + l * 32 + 22]);

			}
			location = most_adv_0_block;
			
			//create FCB for this file
			for (int o = 0; o < 20; o++) {
				fs->volume[4096 + most_adv_0_fcb * 32 + o] = s[o];
			}
			fs->volume[4096 + most_adv_0_fcb * 32 + 20] = G_WRITE;
			fs->volume[4096 + most_adv_0_fcb * 32 + 21] = location / 256;
			fs->volume[4096 + most_adv_0_fcb * 32 + 22] = location % 256;
			fs->volume[4096 + most_adv_0_fcb * 32 + 23] = 0; 
			fs->volume[4096 + most_adv_0_fcb * 32 + 24] = 0;
			fs->volume[4096 + most_adv_0_fcb * 32 + 27] = ctime / 256;
			fs->volume[4096 + most_adv_0_fcb * 32 + 28] = ctime % 256;
			fs->volume[4096 + most_adv_0_fcb * 32 + 29] = UNUSED;
            //update most_adv_0_fcb
			most_adv_0_fcb++;

			


		}
		 
	}

	
	else if (op == G_WRITE) {
		create_time++;
		location = most_adv_0_block;

		//create FCB for this file
		for (int o = 0; o < 20; o++) {
			fs->volume[4096 + most_adv_0_fcb * 32 + o] = s[o];
		}
		fs->volume[4096 + most_adv_0_fcb * 32 + 20] = G_WRITE;
		fs->volume[4096 + most_adv_0_fcb * 32 + 21] = location / 256;
		fs->volume[4096 + most_adv_0_fcb * 32 + 22] = location % 256;
		//printf("location/256: %d; location!256: %d\n", location / 256, location % 256);
		fs->volume[4096 + most_adv_0_fcb * 32 + 23] = 0;
		fs->volume[4096 + most_adv_0_fcb * 32 + 24] = 0;
		fs->volume[4096 + most_adv_0_fcb * 32 + 27] = create_time / 256;
		fs->volume[4096 + most_adv_0_fcb * 32 + 28] = create_time % 256;
		fs->volume[4096 + most_adv_0_fcb * 32 + 29] == UNUSED;
	/*for (int o = 0; o < 20; o++) {
			printf("%c", s[o]);
		}
		printf("create_time: %d\n", create_time);*/
		//update most_adv_0_fcb
		most_adv_0_fcb++;


	}
	return location;
	



}


__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp)
{
	/* Implement read operation here */
	assert(size <= 1024);

	
	
	for (int g = 0; g < size; g++) {
		output[g] = fs->volume[36864 + fp * 32 + g];
	}
	
	
}

__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp)
{
	/* Implement write operation here */
	assert(size <= 1024);
	u16 block_offset = 0;
	if (size % 32 > 0) {

		block_offset = size / 32 + 1;
	}
	else if (size % 32 == 0) {
		block_offset = size / 32;
	}
	if (fs->volume[4096 + (most_adv_0_fcb-1) * 32 + 20] == G_WRITE) {
		//write in storage
		for (int h = 0; h < size; h++) {
			fs->volume[36864 + fp * 32 + h] = input[h];
		}
		//update most_adv_0_block
		most_adv_0_block += block_offset;
		//update modify_time
		modify_time++;
		//update FCB
		fs->volume[4096 + (most_adv_0_fcb - 1) * 32 + 25] = modify_time / 256;
		fs->volume[4096 + (most_adv_0_fcb - 1) * 32 + 26] = modify_time % 256;
		fs->volume[4096 + (most_adv_0_fcb - 1) * 32 + 23] = size / 256;
		fs->volume[4096 + (most_adv_0_fcb - 1) * 32 + 24] = size % 256;
		///printf("size/256: %d; size!256: %d\n", size / 256, size % 256);

	}
	else {
		printf("Wrong operation. Not permitted!\n");
	}
	return fp;

}
__device__ void fs_gsys(FileSystem *fs, int op)
{
	/* Implement LS_D and LS_S operation here */
	
	if (op == LS_D) {
		printf("===sort by modified time===\n");
		u16 largest_mtime = 0;
		int largest = -1;
		int r = 0;
		while (r <= most_adv_0_fcb-1) {
			for (int f = 0; f <= (most_adv_0_fcb - 1); f++) {
				u16 mtime = fs->volume[4096 + f * 32 + 25] * 256 + fs->volume[4096 + f * 32 + 26];
				if (mtime > largest_mtime && fs->volume[4096 + f * 32 + 29]==UNUSED) {
					largest = f;
					largest_mtime = mtime;
					
					
				}
			
			}
			fs->volume[4096 + largest * 32 + 29] = USED;
			/*for (int d = 0; d < 20; d++) {
				printf("%c", fs->volume[4096 + largest * 32 + d]);
			}*/
			printf("%s", &fs->volume[4096 + largest * 32]);
			printf("\n");
			largest_mtime = 0;

			r++;
		}
		for (int f = 0; f <= (most_adv_0_fcb - 1); f++) {
			fs->volume[4096 + f * 32 + 29] = UNUSED;
		} 
	}
	else if (op == LS_S) {
		printf("===sort by size===\n");
		u16 largest_size = 0;
		u16 ctime_of_largest = 0;
		int largest = -1;
		int r = 0;
		while (r <= most_adv_0_fcb - 1) {
			for (int f = 0; f <= (most_adv_0_fcb - 1); f++) {
				//u16 ctime = fs->volume[4096 + f * 32 + 27] * 256 + fs->volume[4096 + f * 32 + 28];
				//u16 ctime_of_largest = fs->volume[4096 + largest * 32 + 27] * 256 + fs->volume[4096 + largest * 32 + 28];
				u16 size = fs->volume[4096 + f * 32 + 23] * 256 + fs->volume[4096 + f * 32 + 24];
				//printf("size is %d\n", size);
				//since files that create later will definitely locate behind files create before
				//condition: ((size > largest_size) || ((size == largest_size) && (ctime < ctime_of_largest)))
				if (fs->volume[4096 + f * 32 + 29] == UNUSED) {
					if (size > largest_size) {
						largest = f;
						largest_size = size;
						ctime_of_largest = fs->volume[4096 + f * 32 + 27] * 256 + fs->volume[4096 + f * 32 + 28];
					}
					else if (size == largest_size) {
					    u16 ctime = fs->volume[4096 + f * 32 + 27] * 256 + fs->volume[4096 + f * 32 + 28];
						if (ctime < ctime_of_largest) {
							largest = f;
							largest_size = size;
							ctime_of_largest = ctime;
						}
					}
					

				}
				
				
			
			}
			//printf("largest is %dth file.\n", largest);
			fs->volume[4096 + largest * 32 + 29] = USED;
			/*for (int d = 0; d < 20; d++) {
				printf("%c", fs->volume[4096 + largest * 32 + d]);
			}*/
			printf("%s", &fs->volume[4096 + largest * 32]);
			printf("  %d\n", largest_size);
			largest_size = 0;
			ctime_of_largest = 0;
			r++;
		}

		for (int f = 0; f <= (most_adv_0_fcb - 1); f++) {
			fs->volume[4096 + f * 32 + 29] = UNUSED;
		}

	}

}

__device__ void fs_gsys(FileSystem *fs, int op, char *s)
{
	/* Implement rm operation here */
	
	if (op == RM) {
		
		bool find = false; 
		int i;
		for (i = 0; i < 1024; i++) {
			for (int j = 0; j < 20; j++) {
				if (fs->volume[4096 + i * 32 + j] == s[j]) {
					if (s[j] == '\0') {
						find = true;
						break;
					}
				}
				else break;
			}
			if (find) {
				break;
			}

		}
		if (!find) {
			printf("file not found! invalid operation.\n");
		}
		else {
			u16 size = fs->volume[4096 + i * 32 + 23] * 256 + fs->volume[4096 + i * 32 + 24];
			u16 block_offset = 0;
			if (size % 32 > 0) {
				block_offset = size / 32+1;
			}
			else if (size % 32 == 0) {
				block_offset = size / 32;
			}
			
			//u16 location_local = fs->volume[4096 + i * 32 + 21] * 256 + fs->volume[4096 + i * 32 + 22];
			// compact FCB
			for (int j = 4096 + (i + 1) * 32; j < 4096 + most_adv_0_fcb * 32; j++) {

				fs->volume[j - 32] = fs->volume[j];


			}
			most_adv_0_fcb--;
			//compact the volume
			u16 original_location = fs->volume[4096 + i * 32 + 21] * 256 + fs->volume[4096 + i * 32 + 22];
			for (int m = 36864 + original_location * 32; m < 36864 + most_adv_0_block * 32; m++) {

				fs->volume[m - block_offset * 32] = fs->volume[m];

			}
			most_adv_0_block -= block_offset;
			// update location in FCB
			int l = i;
			for (l = 4096+i*32; l <= 4096 + (most_adv_0_fcb - 1) * 32; l += 32) {
				u16 new_location = fs->volume[l+21] * 256 + fs->volume[l+22] - block_offset;
				fs->volume[l + 21] = new_location / 256;
				fs->volume[l + 22] = new_location % 256;
			}
			
			


		}

	}
	else {
		printf("permission denied!\n");
	}
}

__device__ void printfcb(FileSystem * fs) {
	
	for (int i = 0; i < most_adv_0_fcb; i++) {
		
		printf("the %dth file: ", i);
		/*for (int j = 0; j < 20; j++) {
			if (fs->volume[4096 + i * 32 + j] == '\0') break;
			printf("%c", fs->volume[4096 + i * 32 + j]);
		}*/
		
			
	    printf("%s", &fs->volume[4096 + i * 32]);
		
		printf(" size: %d", fs->volume[4096 + i * 32 + 23] * 256 + fs->volume[4096 + i * 32 + 24]);
		printf("\n");
	}
}
























































































































			
