﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>


__device__  int count=0;

__device__ void init_invert_page_table(VirtualMemory *vm) {

  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
    vm->invert_page_table[i] = 0x80000000; // invalid := MSB is 1
    vm->invert_page_table[i + vm->PAGE_ENTRIES] = -1; //corresponding virtual address
    vm->invert_page_table[i + 2* vm->PAGE_ENTRIES] = 0;//count number
  }
}

__device__ void vm_init(VirtualMemory *vm, uchar *buffer, uchar *storage,
                        u32 *invert_page_table, int *pagefault_num_ptr,
                        int PAGESIZE, int INVERT_PAGE_TABLE_SIZE,
                        int PHYSICAL_MEM_SIZE, int STORAGE_SIZE,
                        int PAGE_ENTRIES) {
  // init variables
  vm->buffer = buffer;
  vm->storage = storage;
  vm->invert_page_table = invert_page_table;
  vm->pagefault_num_ptr = pagefault_num_ptr;

  // init constants
  vm->PAGESIZE = PAGESIZE;
  vm->INVERT_PAGE_TABLE_SIZE = INVERT_PAGE_TABLE_SIZE;
  vm->PHYSICAL_MEM_SIZE = PHYSICAL_MEM_SIZE;
  vm->STORAGE_SIZE = STORAGE_SIZE;
  vm->PAGE_ENTRIES = PAGE_ENTRIES;

  // before first vm_write or vm_read
  init_invert_page_table(vm);
}

__device__ uchar vm_read(VirtualMemory *vm, u32 addr) {
    int offset = addr % 32; //capture last 5 digits
    uchar result;
    bool find = false;
    for (int i = 0; i < vm->PAGE_ENTRIES; i++)
    {
        

        if (vm->invert_page_table[i + vm->PAGE_ENTRIES] == addr>>5 && vm->invert_page_table[i]==0) //2^10 -> 2^12
        {
            //printf("read successful, virtual address is %d!\n", vm->invert_page_table[i + vm->PAGE_ENTRIES]);
            result = vm->buffer[i*32 + offset];//2^15
            count++;
            vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES]=count;//count plus
            find = true;
            break;
        }
    }
    if (!find)
    {
        (*vm->pagefault_num_ptr)++; //increase page fault number
        //printf("preform read LRU\n");
        int comp_count = 999999999999;
        int smallest = 0;
        for (int i = 0; i < vm->PAGE_ENTRIES; i++)
        {
            if (vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES] < comp_count)
            {
                comp_count = vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES];
                smallest = i;
            }
        }
        int virtual_page_number = vm->invert_page_table[smallest + vm->PAGE_ENTRIES];//2^12
        for (int j = 0; j < vm->PAGESIZE; j++)
        {
            int offset = j;
            u32 physical_address = smallest * 32 + offset;
            u32 virtual_address = virtual_page_number * 32 + offset;
            vm->storage[virtual_address] = vm->buffer[physical_address]; //swap out to storage
        }
        vm->invert_page_table[smallest + vm->PAGE_ENTRIES] = addr >> 5; //update invert page table
        count++;
        vm->invert_page_table[smallest + 2 * vm->PAGE_ENTRIES] = count; //update count
        for (int j = 0; j < vm->PAGESIZE; j++)
        {
            int offset = j;
            int virtual_page_number = addr>>5;
            u32 virtual_address = virtual_page_number * 32 + offset;
            u32 physical_address = smallest * 32 + offset;
            vm->buffer[physical_address] = vm->storage[virtual_address]; //swap the desired page in physical memory
        }
        //printf("virtual page %d is replaced by new virtual page %d", virtual_page_number, addr >> 5);
        result = vm->buffer[smallest * 32 + offset];

    }



  /* Complete vm_read function to read single element from data buffer */
  return result; //TODO
}

__device__ void vm_write(VirtualMemory *vm, u32 addr, uchar value) {
  /* Complete vm_write function to write value into data buffer */
    
    
    int virtual_page_number = addr >> 5;
    int offset = addr % 32;
    //printf("vpn = %d, offset = %d\n", virtual_page_number, offset);
    bool find=false;

    for (int i = 0; i < vm->PAGE_ENTRIES; i++)
    {
        if (vm->invert_page_table[i + vm->PAGE_ENTRIES] == virtual_page_number)
        {
            find = true; //if find
            u32 physical_address = i * 32 + offset;
            //printf("physical addr: %d\n", physical_address);
            //vm->invert_page_table[i] = 0; //set to not empty
            //vm->invert_page_table[i + vm->PAGE_ENTRIES] = virtual_page_number;
            vm->buffer[physical_address] = value; //write in physical memory
            count++;//more recently written page will have bigger count, so count is static int variable
            vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES] = count;//update count
            break; //once find, then stop
        }
        
    }
    if (!find)
    {
        (*vm->pagefault_num_ptr)++;
        //printf("This page faault number = %d\n", *(vm->pagefault_num_ptr));
        for (int i = 0; i < vm->PAGE_ENTRIES; i++)
        {
            if (vm->invert_page_table[i] == 0x80000000)
            {
                find = true; //if find
                u32 physical_address = i << 5 + offset;
                vm->invert_page_table[i] = 0; //set to not empty
                vm->invert_page_table[i + vm->PAGE_ENTRIES] = virtual_page_number;
                //printf("bind virtual page number %d to physiacal frame number %d\n", virtual_page_number, i);
                vm->buffer[physical_address] = value; //write in physical memory
                count++;//more recently written page will have bigger count, so count is static int variable
                vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES] = count;//update count
                break; //once find, then stop

            }

        }
    }
    if(!find)
    {
        //if there is no physical page for this logical page, and there is no empty page, there is need to swap
        // (*vm->pagefault_num_ptr)++;
        //printf("perform LRU\n");
        // printf("This page faault number = %d\n", *(vm->pagefault_num_ptr));
        int comp_count = 999999999999;
        int smallest = 0;
        for (int i = 0; i < vm->PAGE_ENTRIES; i++)
        {
            if (vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES] < comp_count)
            {
                comp_count = vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES];
                smallest = i;
            }
        }
        //printf("smallest virtual page number %d replaced by new %d\n", smallest, virtual_page_number);




        for (int j = 0; j < vm->PAGESIZE; j++)
        {
            int offset_local = j;
            u32 physical_address = smallest * 32 + offset_local;
            u32 virtual_address = vm->invert_page_table[smallest + vm->PAGE_ENTRIES] * 32 + offset_local;
            vm->storage[virtual_address] = vm->buffer[physical_address]; //swap out to storage
        }


        vm->invert_page_table[smallest + vm->PAGE_ENTRIES] = virtual_page_number; //update invert page table
        count++;
        vm->invert_page_table[smallest + 2 * vm->PAGE_ENTRIES] = count; //update count
        u32 physical_address = smallest * 32 + offset;
        vm->buffer[physical_address] = value;
    }


    
    
   
    /*if (offset == 0)
    {
        bool find = false;
        for (int i = 0; i < vm->PAGE_ENTRIES; i++)
        {
            if (vm->invert_page_table[i] == 0x80000000)
            {
                find = true; //if find
                u32 physical_address = i << 5 + offset;
                vm->invert_page_table[i] = 0; //set to not empty
                vm->invert_page_table[i + vm->PAGE_ENTRIES] = virtual_page_number;
                vm->buffer[physical_address] = value; //write in physical memory
                count++;//more recently written page will have bigger count, so count is static int variable
                vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES] = count;//update count
                break; //once find, then stop

            }
        }
        if (!find)
        {
            (*vm->pagefault_num_ptr)++;
            int comp_count = 999999999999;
            int smallest = 0;
            for (int i = 0; i < vm->PAGE_ENTRIES; i++)
            {
                if (vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES] < comp_count)
                {
                    comp_count = vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES];
                    smallest = i;
                }
            }
            


            
           
            for (int j = 0; j < vm->PAGESIZE; j++)
            {
                int offset = j;
                u32 physical_address = smallest << 5 + offset;
                u32 virtual_address = vm->invert_page_table[smallest + vm->PAGE_ENTRIES] << 5 + offset;
                vm->storage[virtual_address] = vm->buffer[physical_address]; //swap out to storage
            }
            

            vm->invert_page_table[smallest + vm->PAGE_ENTRIES] = virtual_page_number; //update invert page table
            count++;
            vm->invert_page_table[smallest + 2 * vm->PAGE_ENTRIES] = count; //update count
            u32 physical_address = smallest << 5 + offset;
            vm->buffer[physical_address] = value;

        }


    }
    else
    {
        bool find=false;
        for (int i = 0; i < vm->PAGE_ENTRIES; i++)
        {
            if (vm->invert_page_table[i + vm->PAGE_ENTRIES] == virtual_page_number)
            {
                find=true;
                u32 physical_address = i << 5 + offset;
                vm->buffer[physical_address] = value;
                count++;
                vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES] = count;

                break;
            }
        }
        if (!find)
        {
            (*vm->pagefault_num_ptr)++;
            int comp_count = 999999999999;
            int smallest = 0;
            for (int i = 0; i < vm->PAGE_ENTRIES; i++)
            {
                if (vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES] < comp_count)
                {
                    comp_count = vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES];
                    smallest = i;
                }
            }
            for (int j = 0; j < vm->PAGESIZE; j++)
            {
                int offset = j;
                u32 physical_address = smallest << 5 + offset;
                u32 virtual_address = vm->invert_page_table[smallest + vm->PAGE_ENTRIES] << 5 + offset;
                vm->storage[virtual_address] = vm->buffer[physical_address]; //swap out to storage
            }


            vm->invert_page_table[smallest + vm->PAGE_ENTRIES] = virtual_page_number; //update invert page table
            count++;
            vm->invert_page_table[smallest + 2 * vm->PAGE_ENTRIES] = count; //update count
            u32 physical_address = smallest << 5 + offset;
            vm->buffer[physical_address] = value;

        }
    }*/
}

__device__ void vm_snapshot(VirtualMemory* vm, uchar* results, int offset,
    int input_size)

{
    uchar result;
    for (int i = 0; i < input_size; ++i) {
        result = vm_read(vm, i);
        results[i] = result;
    }
    
}
   

  /* Complete snapshot function togther with vm_read to load elements from data
   * to result buffer */


